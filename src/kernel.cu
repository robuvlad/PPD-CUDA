#include <stdio.h>
#include <hip/hip_runtime.h>

//TESTING

#define SIZE 1024

__global__ void VectorAdd(int *a, int *b, int *c, int n) { //global -> tells the compiler that this function will be executed on the gpu

	int i = threadIdx.x;

	if (i < n)
		c[i] = a[i] + b[i];
}

int main() {
	int *a, *b, *c;

	hipMallocManaged(&a, SIZE * sizeof(int)); // cudaMallocManaged returns a pointer
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; i++) {
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	VectorAdd <<<1, SIZE>>> (a, b, c, SIZE); // <<<no of thread blocks, no of threads within each thread block>>>

	hipDeviceSynchronize(); // cpu waits for the kernels to complete before continuing

	for (int i = 0; i < 10; i++) {
		printf("c[%d] = %d\n", i, c[i]);
	}

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}